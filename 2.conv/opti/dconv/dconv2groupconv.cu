/*
 * dgrad conv 转换成 stride_h * stride_w 组 stride = 1 的 dgrad conv
 */

#include <iostream>
#include <cudnn.h>
#include <hip/hip_runtime.h>

// #define N 6
// #define IN_C 5
// #define IN_H 7
// #define IN_W 7
// #define K_H 3
// #define K_W 3
// #define OUT_C 3
// #define OUT_H 3
// #define OUT_W 3
// #define PAD_H 1
// #define PAD_W 1
// #define STRIDE_H 2
// #define STRIDE_W 2
// #define DILATION_H 2
// #define DILATION_W 2

#define N 1
#define IN_C 1
#define IN_H 3
#define IN_W 3
#define K_H 2
#define K_W 2
#define OUT_C 1
#define OUT_H 2
#define OUT_W 2
#define PAD_H 0
#define PAD_W 0
#define STRIDE_H 1
#define STRIDE_W 1
#define DILATION_H 1
#define DILATION_W 1

// void rand_data(float *data, int num, float min, float max) {
//     for (int i = 0; i < num; i++) {
//         data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() / (float)RAND_MAX) + min);
//     }
// }
void rand_data(float *data, int num, float min, float max) {
    for (int i = 0; i < num; i++) {
        data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() % 5) + min);
    }
}

void dgrad(const float *dy, const float *w, float *dx) {

#define NEW_K_H ((K_H - 1) * (DILATION_H - 1) + K_H)
#define NEW_K_W ((K_W - 1) * (DILATION_W - 1) + K_W)
#define STRIDE_K_H (NEW_K_H + STRIDE_H - 1) / STRIDE_H
#define STRIDE_K_W (NEW_K_W + STRIDE_W - 1) / STRIDE_W
#define STRIDE_IN_H (IN_H + STRIDE_H - 1) / STRIDE_H
#define STRIDE_IN_W (IN_W + STRIDE_W - 1) / STRIDE_W

    for (int s_h = 0; s_h < STRIDE_H; ++s_h) {
        for (int s_w = 0; s_w < STRIDE_W; ++s_w) {

            float *wh = (float *)malloc(OUT_C * IN_C * STRIDE_K_H * STRIDE_K_W * sizeof(float));
            float *dxh = (float *)malloc(N * IN_C * STRIDE_IN_H * STRIDE_IN_W * sizeof(float));
            memset(wh, 0, OUT_C * IN_C * STRIDE_K_H * STRIDE_K_W * sizeof(float));
            memset(dxh, 0, N * IN_C * STRIDE_IN_H * STRIDE_IN_W * sizeof(float));
            // 1. dy
            // 2. (sh,sw)对应的w
            for (int out_c = 0; out_c < OUT_C; ++out_c) {
                for (int in_c = 0; in_c < IN_C; ++in_c) {
                    for (int k_h = 0; k_h < STRIDE_K_H; ++k_h) {
                        for (int k_w = 0; k_w < STRIDE_K_W; ++k_w) {
                            if ((k_h * STRIDE_H + s_h) % DILATION_H == 0 && (k_w * STRIDE_W + s_w) % DILATION_W == 0) {
                                int real_k_h = (k_h * STRIDE_H + s_h) / DILATION_H;
                                int real_k_w = (k_w * STRIDE_W + s_w) / DILATION_W;
                                int pos1 = ((out_c * IN_C + in_c) * STRIDE_K_H + k_h) * STRIDE_K_W + k_w;
                                int pos2 = ((out_c * IN_C + in_c) * K_H + real_k_h) * K_W + real_k_w;
                                wh[pos1] = w[pos2];
                            } 
                        }
                    }
                }
            }
    printf("\n");
    for (int i = 0; i < STRIDE_K_H; ++i) {
        for (int j = 0; j < STRIDE_K_W; ++j) {
            printf("%3.0f ", wh[i*STRIDE_K_W+j]);
        }
        printf("\n");
    }
    printf("\n");
            // 3. conv -> (sh,sw)对应的dx
            for (int n = 0; n < N; ++n) {
                for (int in_c = 0; in_c < IN_C; ++in_c) {
                    for (int in_h = 0; in_h < STRIDE_IN_H; ++in_h) {
                        for (int in_w = 0; in_w < STRIDE_IN_W; ++in_w) {
                            float temp = 0.0f;
                            for (int k_h = 0; k_h < STRIDE_K_H; ++k_h) {
                                for (int k_w = 0; k_w < STRIDE_K_W; ++k_w) {
                                    int out_h = in_h + k_h;
                                    int out_w = in_w + k_w;
                                    for (int out_c = 0; out_c < OUT_C; ++out_c) {
                                        int dypos = (((n * OUT_C) + out_c) * OUT_H + out_h) *  OUT_W + out_w;
                                        int wpos = (((out_c * IN_C) + in_c) * STRIDE_K_H + k_h) *  STRIDE_K_W + k_w;
                                        printf("%3.0f %3.0f ", dy[dypos], w[wpos]);
                                        temp += dy[dypos] * w[wpos];
                                    }
                                }
                            }
                            printf("\n");
                            int real_in_h = in_h * STRIDE_H + s_h - PAD_H;
                            int real_in_w = in_w * STRIDE_W + s_w - PAD_W;
                            if (real_in_h >= 0 && real_in_h < IN_H && real_in_w >= 0 && real_in_w < IN_W) {
                                int dxpos = (((n * IN_C) + in_c) * IN_H + real_in_h) * IN_W + real_in_w;
                                dx[dxpos] = temp;
                            }
                        }
                    }
                }
            }
            // 4. write back

            free(wh);
            free(dxh);

        }
    }

}

int main() {

    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Initialize cuDNN
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    // Define the dy tensor descriptor
    cudnnTensorDescriptor_t dy_desc;
    cudnnCreateTensorDescriptor(&dy_desc);
    cudnnSetTensor4dDescriptor(dy_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, OUT_C, OUT_H, OUT_W);

    // Define the w descriptor
    cudnnFilterDescriptor_t w_desc;
    cudnnCreateFilterDescriptor(&w_desc);
    cudnnSetFilter4dDescriptor(w_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, OUT_C, IN_C, K_H, K_W);

    // Define the dx tensor descriptor
    cudnnTensorDescriptor_t dx_desc;
    cudnnCreateTensorDescriptor(&dx_desc);
    cudnnSetTensor4dDescriptor(dx_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, IN_C, IN_H, IN_W);

    // Define the convolution descriptor
    cudnnConvolutionDescriptor_t conv_desc;
    cudnnCreateConvolutionDescriptor(&conv_desc);
    cudnnSetConvolution2dDescriptor(conv_desc, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);

    int size_dy = N * OUT_C * OUT_H * OUT_W;
    int size_w = OUT_C * IN_C * K_H * K_W;
    int size_dx = N * IN_C * IN_H * IN_W;

    // Allocate host memory
    float *h_dy, *h_w, *h_dx;
    h_dy = (float*)malloc(size_dy * sizeof(float));
    h_w = (float*)malloc(size_w * sizeof(float));
    h_dx = (float*)malloc(size_dx * sizeof(float));

    // Initialization
    rand_data(h_dy, size_dy, -1, 1);
    rand_data(h_w, size_w, -1, 1);
    rand_data(h_dx, size_dx, 0, 0);

    // Allocate device memory
    float *d_dy, *d_w, *d_dx;
    hipMalloc((void**)&d_dy, size_dy * sizeof(float));
    hipMalloc((void**)&d_w, size_w * sizeof(float));
    hipMalloc((void**)&d_dx, size_dx * sizeof(float));

    // Memcpy: host -> device
    hipMemcpy(d_dy, h_dy, size_dy * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_dx, h_dx, size_dx * sizeof(float), hipMemcpyHostToDevice);

    // Perform bgrad convolution
    float alpha = 1.0f, beta = 0.0f;
    cudnnConvolutionBackwardData(cudnn, &alpha, w_desc, d_w, dy_desc, d_dy, conv_desc, CUDNN_CONVOLUTION_BWD_DATA_ALGO_0, nullptr, 0, &beta, dx_desc, d_dx);

    // Memcpy: device -> host
    hipMemcpy(h_dx, d_dx, size_dx * sizeof(float), hipMemcpyDeviceToHost);
    
    // Compare
    float *calc_dx = (float*)malloc(size_dx * sizeof(float));
    rand_data(calc_dx, size_dx, 0, 0);
    for (int i = 0; i < OUT_H; ++i) {
        for (int j = 0; j < OUT_W; ++j) {
            printf("%3.0f ", h_dy[i*OUT_W+j]);
        }
        printf("\n");
    }
    printf("\n");
    for (int i = 0; i < K_H; ++i) {
        for (int j = 0; j < K_W; ++j) {
            printf("%3.0f ", h_w[(K_H-i-1)*K_W+(K_W-j-1)]);
        }
        printf("\n");
    }
    printf("\n");
    dgrad(h_dy, h_w, calc_dx);
    float diff = 0.0f;
    for (int i = 0; i < size_dx; ++i) {
        printf("\n--------diff:%3.0f %3.0f------\n", h_dx[i], calc_dx[i]);
        diff += (h_dx[i] - calc_dx[i]);
    }
    printf("\n--------diff:%f------\n", diff);

    // Clean up
    free(h_dy);
    free(h_w);
    free(h_dx);
    free(calc_dx);
    hipFree(d_dy);
    hipFree(d_w);
    hipFree(d_dx);
    cudnnDestroyTensorDescriptor(dy_desc);
    cudnnDestroyFilterDescriptor(w_desc);
    cudnnDestroyTensorDescriptor(dx_desc);
    cudnnDestroyConvolutionDescriptor(conv_desc);
    cudnnDestroy(cudnn);

    return 0;
}
