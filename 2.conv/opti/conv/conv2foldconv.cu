/*
 * conv抽取H,W维补充C维 (限制p=0,d=1)
 * 举例如下，抽W维补充Cin维
 */

#include <iostream>
#include <cudnn.h>
#include <hip/hip_runtime.h>

#define GROUP 2
#define N 6
#define IN_C 5
#define IN_H 7
#define IN_W 7
#define K_H 3
#define K_W 3
#define OUT_C 3
#define OUT_H 3
#define OUT_W 3
#define PAD_H 0
#define PAD_W 0
#define STRIDE_H 2
#define STRIDE_W 2
#define DILATION_H 1
#define DILATION_W 1

void rand_data(float *data, int num, float min, float max) {
    for (int i = 0; i < num; i++) {
        data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() / (float)RAND_MAX) + min);
    }
}

void conv(const float* x, const float* w, float* y) {
    
    float *xh = (float *)malloc(N * IN_C * GROUP * IN_H * (IN_W + GROUP - 1) / GROUP * sizeof(float));
    float *wh = (float *)malloc(OUT_C * IN_C * GROUP * K_H * (K_W + GROUP - 1) / GROUP * sizeof(float));
    memset(xh, 0, N * IN_C * GROUP * IN_H * (IN_W + GROUP - 1) / GROUP * sizeof(float));
    memset(wh, 0, OUT_C * IN_C * GROUP * K_H * (K_W + GROUP - 1) / GROUP * sizeof(float));
    // 1. 转换x (N,Cin,Hin,Win) -> (N,Cin*GROUP,Hin,Win/GROUP)
    for (int n = 0; n < N; ++n) {
        for (int in_c = 0; in_c < IN_C; ++in_c) {
            for (int in_h = 0; in_h < IN_H; ++in_h) {
                for (int in_w = 0; in_w < IN_W; ++in_w) {
                    int real_in_c = (in_w % GROUP) * IN_C + in_c;
                    int real_in_w = in_w / GROUP;
                    int pos1 = (((n) * IN_C * GROUP + real_in_c) * IN_H + in_h) * (IN_W + GROUP - 1) / GROUP + real_in_w;
                    int pos2 = (((n) * IN_C + in_c) * IN_H + in_h) * IN_W + in_w;
                    xh[pos1] = x[pos2];
                }
            }
        }
    }
    // 2. 转换w (Cout,Cin,Hk,Wk) -> (Cout,Cin \times 2,Hk,Wk/2)
    for (int out_c = 0; out_c < OUT_C; ++out_c) {
        for (int in_c = 0; in_c < IN_C; ++in_c) {
            for (int k_h = 0; k_h < K_H; ++k_h) {
                for (int k_w = 0; k_w < K_W; ++k_w) {
                    int real_in_c = (k_w % GROUP) * IN_C + in_c;
                    int real_k_w = k_w / GROUP;
                    int pos1 = (((out_c) * IN_C * GROUP + real_in_c) * K_H + k_h) * (K_W + GROUP - 1) / GROUP + real_k_w;
                    int pos2 = (((out_c) * IN_C + in_c) * K_H + k_h) * K_W + k_w;
                    wh[pos1] = w[pos2];
                }
            }
        }
    }
    // 3. conv
    for (int n = 0; n < N; ++n) {
        for (int out_c = 0; out_c < OUT_C; ++out_c) {
            for (int out_h = 0; out_h < OUT_H; ++out_h) {
                for (int out_w = 0; out_w < OUT_W; ++out_w) {
                    float temp = 0.0f;
                    for (int k_h = 0; k_h < K_H; ++k_h) {
                        for (int k_w = 0; k_w < (K_W + GROUP - 1) / GROUP; ++k_w) {
                            int real_in_h = out_h * STRIDE_H + k_h;
                            int real_in_w = out_w + k_w;
                            if (real_in_h >= 0 && real_in_h < IN_H && real_in_w >= 0 && real_in_w < (IN_W + GROUP - 1) / GROUP) {
                                int real_k_h = k_h;
                                int real_k_w = k_w;
                                for (int in_c = 0; in_c < IN_C * GROUP; ++in_c) {
                                    int xpos = (((n) * IN_C * GROUP + in_c) * IN_H + real_in_h) * (IN_W + GROUP - 1) / GROUP + real_in_w;
                                    int wpos = (((out_c) * IN_C * GROUP + in_c) * K_H + real_k_h) * (K_W + GROUP - 1) / GROUP + real_k_w;
                                    temp += (float)xh[xpos] * (float)wh[wpos];
                                }
                            }
                        }
                    }
                    // 4.累加
                    int ypos = (((n) * OUT_C + out_c) * OUT_H + out_h) * OUT_W + out_w;
                    y[ypos] += temp;
                }
            }
        }
    }

    free(xh);
    free(wh);

}

int main() {
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Initialize cuDNN
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    // Define the x tensor descriptor
    cudnnTensorDescriptor_t x_desc;
    cudnnCreateTensorDescriptor(&x_desc);
    cudnnSetTensor4dDescriptor(x_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, IN_C, IN_H, IN_W);

    // Define the convolution descriptor
    cudnnFilterDescriptor_t w_desc;
    cudnnCreateFilterDescriptor(&w_desc);
    cudnnSetFilter4dDescriptor(w_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, OUT_C, IN_C, K_H, K_W);

    // Define the convolution descriptor
    cudnnConvolutionDescriptor_t conv_desc;
    cudnnCreateConvolutionDescriptor(&conv_desc);
    cudnnSetConvolution2dDescriptor(conv_desc, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);

    // Define the y tensor descriptor
    cudnnTensorDescriptor_t y_desc;
    cudnnCreateTensorDescriptor(&y_desc);
    cudnnSetTensor4dDescriptor(y_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, OUT_C, OUT_H, OUT_W);

    int size_x = N * IN_C * IN_H * IN_W;
    int size_w = OUT_C * IN_C * K_H * K_W;
    int size_y = N * OUT_C * OUT_H * OUT_W;

    // Allocate host memory for x, w, and y
    float *h_x, *h_y, *h_w;
    h_x = (float*)malloc(size_x * sizeof(float));
    h_w = (float*)malloc(size_w * sizeof(float));
    h_y = (float*)malloc(size_y * sizeof(float));

    // Initialization
    rand_data(h_x, size_x, -1, 1);
    rand_data(h_w, size_w, -1, 1);
    rand_data(h_y, size_y, 0, 0);

    // Allocate device memory for x, w, and y
    float *d_x, *d_y, *d_w;
    hipMalloc((void**)&d_x, size_x * sizeof(float));
    hipMalloc((void**)&d_w, size_w * sizeof(float));
    hipMalloc((void**)&d_y, size_y * sizeof(float));

    // Memcpy: host -> device
    hipMemcpy(d_x, h_x, size_x * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size_y * sizeof(float), hipMemcpyHostToDevice);

    // Perform forward convolution
    float alpha = 1.0f, beta = 0.0f;
    cudnnConvolutionForward(cudnn, &alpha, x_desc, d_x, w_desc, d_w, conv_desc, CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, y_desc, d_y);

    // Memcpy: device -> host
    hipMemcpy(h_y, d_y, size_y * sizeof(float), hipMemcpyDeviceToHost);

    // Compare
    float *calc_y = (float*)malloc(size_y * sizeof(float));
    rand_data(calc_y, size_y, 0, 0);
    conv(h_x, h_w, calc_y);
    float diff = 0.0f;
    for (int i = 0; i < size_y; ++i) {
        diff += (h_y[i] - calc_y[i]);
    }
    printf("\n--------diff:%f------\n", diff);

    // Clean up
    free(h_x);
    free(h_w);
    free(h_y);
    free(calc_y);
    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_y);
    cudnnDestroyTensorDescriptor(x_desc);
    cudnnDestroyFilterDescriptor(w_desc);
    cudnnDestroyTensorDescriptor(y_desc);
    cudnnDestroyConvolutionDescriptor(conv_desc);
    cudnnDestroy(cudnn);

    return 0;
}
