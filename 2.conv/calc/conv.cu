#include "hip/hip_runtime.h"
#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
// #include "./slidingwindow.h"
// #include "./img2col.h"
#include "./fft.h"

// #define N 6
// #define IN_C 5
// #define IN_H 7
// #define IN_W 7
// #define K_H 3
// #define K_W 3
// #define OUT_C 3
// #define OUT_H 3
// #define OUT_W 3
// #define PAD_H 1
// #define PAD_W 1
// #define STRIDE_H 2
// #define STRIDE_W 2
// #define DILATION_H 2
// #define DILATION_W 2

// fft存在限制，pad=0，stride=1，dilation=1
#define N 6
#define IN_C 5
#define IN_H 7
#define IN_W 7
#define K_H 3
#define K_W 3
#define OUT_C 3
#define OUT_H 5
#define OUT_W 5
#define PAD_H 0
#define PAD_W 0
#define STRIDE_H 1
#define STRIDE_W 1
#define DILATION_H 1
#define DILATION_W 1

void rand_data(float *data, int num, float min, float max) {
    for (int i = 0; i < num; i++) {
        data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() / (float)RAND_MAX) + min);
    }
}

int main() {
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Define the x tensor descriptor
    hipdnnTensorDescriptor_t x_desc;
    hipdnnCreateTensorDescriptor(&x_desc);
    hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, IN_C, IN_H, IN_W);

    // Define the convolution descriptor
    hipdnnFilterDescriptor_t w_desc;
    hipdnnCreateFilterDescriptor(&w_desc);
    hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OUT_C, IN_C, K_H, K_W);

    // Define the convolution descriptor
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // Define the y tensor descriptor
    hipdnnTensorDescriptor_t y_desc;
    hipdnnCreateTensorDescriptor(&y_desc);
    hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, OUT_C, OUT_H, OUT_W);

    int size_x = N * IN_C * IN_H * IN_W;
    int size_w = OUT_C * IN_C * K_H * K_W;
    int size_y = N * OUT_C * OUT_H * OUT_W;

    // Allocate host memory for x, w, and y
    float *h_x, *h_y, *h_w;
    h_x = (float*)malloc(size_x * sizeof(float));
    h_w = (float*)malloc(size_w * sizeof(float));
    h_y = (float*)malloc(size_y * sizeof(float));

    // Initialization
    rand_data(h_x, size_x, -1, 1);
    rand_data(h_w, size_w, -1, 1);
    rand_data(h_y, size_y, 0, 0);

    // Allocate device memory for x, w, and y
    float *d_x, *d_y, *d_w;
    hipMalloc((void**)&d_x, size_x * sizeof(float));
    hipMalloc((void**)&d_w, size_w * sizeof(float));
    hipMalloc((void**)&d_y, size_y * sizeof(float));

    // Memcpy: host -> device
    hipMemcpy(d_x, h_x, size_x * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size_y * sizeof(float), hipMemcpyHostToDevice);

    // Perform forward convolution
    float alpha = 1.0f, beta = 0.0f;
    hipdnnConvolutionForward(cudnn, &alpha, x_desc, d_x, w_desc, d_w, conv_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, y_desc, d_y);

    // Memcpy: device -> host
    hipMemcpy(h_y, d_y, size_y * sizeof(float), hipMemcpyDeviceToHost);

    // Compare
    float *calc_y = (float*)malloc(size_y * sizeof(float));
    memset(calc_y, 0, size_y * sizeof(float));
    // slidingwindow(h_x, h_w, calc_y, N, IN_C, IN_H, IN_W, K_H, K_W, OUT_C, OUT_H, OUT_W, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W);
    // img2col(h_x, h_w, calc_y, N, IN_C, IN_H, IN_W, K_H, K_W, OUT_C, OUT_H, OUT_W, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W);
    convfft(h_x, h_w, calc_y, N, IN_C, IN_H, IN_W, K_H, K_W, OUT_C, OUT_H, OUT_W, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W);

    float diff = 0.0f;
    for (int i = 0; i < size_y; ++i) {
        diff += (h_y[i] - calc_y[i]);
    }
    printf("\n--------diff:%f------\n", diff);

    // Clean up
    free(h_x);
    free(h_w);
    free(h_y);
    free(calc_y);
    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_y);
    hipdnnDestroyTensorDescriptor(x_desc);
    hipdnnDestroyFilterDescriptor(w_desc);
    hipdnnDestroyTensorDescriptor(y_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);

    return 0;
}
