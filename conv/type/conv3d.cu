#include <iostream>
#include <cudnn.h>
#include <hip/hip_runtime.h>

#define N 6
#define IN_C 5
#define IN_D 7
#define IN_H 7
#define IN_W 7
#define K_D 3
#define K_H 3
#define K_W 3
#define OUT_C 3
#define OUT_D 3
#define OUT_H 3
#define OUT_W 3
#define PAD_D 2
#define PAD_H 1
#define PAD_W 1
#define STRIDE_D 2
#define STRIDE_H 2
#define STRIDE_W 2
#define DILATION_D 3
#define DILATION_H 2
#define DILATION_W 2

void rand_data(float *data, int num, float min, float max) {
    for (int i = 0; i < num; i++) {
        data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() / (float)RAND_MAX) + min);
    }
}

void conv3d(const float* x, const float* w, float* y) {

#define IX(n, in_c, in_d, in_h, in_w) (((((n) * IN_C + in_c) * IN_D + in_d) * IN_H + in_h) * IN_W + in_w)
#define IW(out_c, in_c, k_d, k_h, k_w) (((((out_c) * IN_C + in_c) * K_D + k_d) * K_H + k_h) * K_W + k_w)
#define IY(n, out_c, out_d, out_h, out_w) (((((n) * OUT_C + out_c) * OUT_D + out_d) * OUT_H + out_h) * OUT_W + out_w)

    for (int n = 0; n < N; ++n) {
        for (int out_c = 0; out_c < OUT_C; ++out_c) {
            for (int out_d = 0; out_d < OUT_D; ++out_d) {
                for (int out_h = 0; out_h < OUT_H; ++out_h) {
                    for (int out_w = 0; out_w < OUT_W; ++out_w) {
                        float temp = 0.0f;
                        for (int k_d = 0; k_d < (DILATION_D - 1) * (K_D - 1) + K_D; k_d += DILATION_D) {
                            for (int k_h = 0; k_h < (DILATION_H - 1) * (K_H - 1) + K_H; k_h += DILATION_H) {
                                for (int k_w = 0; k_w < (DILATION_W - 1) * (K_W - 1) + K_W; k_w += DILATION_W) {
                                    int real_in_d = out_d * STRIDE_D + k_d - PAD_D;
                                    int real_in_h = out_h * STRIDE_H + k_h - PAD_H;
                                    int real_in_w = out_w * STRIDE_W + k_w - PAD_W;
                                    if (real_in_d >= 0 && real_in_d < IN_D && real_in_h >= 0 && real_in_h < IN_H && real_in_w >= 0 && real_in_w < IN_W) {
                                        int real_k_d = k_d / DILATION_D;
                                        int real_k_h = k_h / DILATION_H;
                                        int real_k_w = k_w / DILATION_W;
                                        for (int in_c = 0; in_c < IN_C; ++in_c) {
                                            temp += (float)x[IX(n, in_c, real_in_d, real_in_h, real_in_w)] * (float)w[IW(out_c, in_c, real_k_d, real_k_h, real_k_w)];
                                        }
                                    }
                                }
                            }
                        }
                        y[IY(n, out_c, out_d, out_h, out_w)] = temp;
                    }
                }
            }
        }
    }

}

int main() {
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Initialize cuDNN
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    // Define the x tensor descriptor
    cudnnTensorDescriptor_t x_desc;
    cudnnCreateTensorDescriptor(&x_desc);
    const int x_dim[] = {N, IN_C, IN_D, IN_H, IN_W};
    const int x_stride[] = {x_dim[1] * x_dim[2] * x_dim[3] * x_dim[4], x_dim[2] * x_dim[3] * x_dim[4], x_dim[3] * x_dim[4], x_dim[4], 1};
    cudnnSetTensorNdDescriptor(x_desc, CUDNN_DATA_FLOAT, 5, x_dim, x_stride);

    // Define the convolution descriptor
    cudnnFilterDescriptor_t w_desc;
    cudnnCreateFilterDescriptor(&w_desc);
    const int w_dim[] = {OUT_C, IN_C, K_D, K_H, K_W};
    cudnnSetFilterNdDescriptor(w_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 5, w_dim);

    // Define the convolution descriptor
    cudnnConvolutionDescriptor_t conv_desc;
    cudnnCreateConvolutionDescriptor(&conv_desc);
    const int PAD[] = {PAD_D, PAD_H, PAD_W};
    const int STRIDE[] = {STRIDE_D, STRIDE_H, STRIDE_W};
    const int DILATION[] = {DILATION_D, DILATION_H, DILATION_W};
    cudnnSetConvolutionNdDescriptor(conv_desc, 3, PAD, STRIDE, DILATION, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);

    // Define the y tensor descriptor
    cudnnTensorDescriptor_t y_desc;
    cudnnCreateTensorDescriptor(&y_desc);
    const int y_dim[] = {N, OUT_C, OUT_D, OUT_H, OUT_W};
    const int y_stride[] = {y_dim[1] * y_dim[2] * y_dim[3] * y_dim[4], y_dim[2] * y_dim[3] * y_dim[4], y_dim[3] * y_dim[4], y_dim[4], 1};
    cudnnSetTensorNdDescriptor(y_desc, CUDNN_DATA_FLOAT, 5, y_dim, y_stride);

    int size_x = N * IN_C * IN_D * IN_H * IN_W;
    int size_w = OUT_C * IN_C * K_D * K_H * K_W;
    int size_y = N * OUT_C * OUT_D * OUT_H * OUT_W;

    // Allocate host memory for x, w, and y
    float *h_x, *h_y, *h_w;
    h_x = (float*)malloc(size_x * sizeof(float));
    h_w = (float*)malloc(size_w * sizeof(float));
    h_y = (float*)malloc(size_y * sizeof(float));

    // Initialization
    rand_data(h_x, size_x, -1, 1);
    rand_data(h_w, size_w, -1, 1);
    rand_data(h_y, size_y, 0, 0);

    // Allocate device memory for x, w, and y
    float *d_x, *d_y, *d_w;
    hipMalloc((void**)&d_x, size_x * sizeof(float));
    hipMalloc((void**)&d_w, size_w * sizeof(float));
    hipMalloc((void**)&d_y, size_y * sizeof(float));

    // Memcpy: host -> device
    hipMemcpy(d_x, h_x, size_x * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size_y * sizeof(float), hipMemcpyHostToDevice);

    // Perform forward convolution
    float alpha = 1.0f, beta = 0.0f;
    cudnnStatus_t res = cudnnConvolutionForward(cudnn, &alpha, x_desc, d_x, w_desc, d_w, conv_desc, CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, y_desc, d_y);

    // Memcpy: device -> host
    hipMemcpy(h_y, d_y, size_y * sizeof(float), hipMemcpyDeviceToHost);

    // Compare
    float *calc_y = (float*)malloc(size_y * sizeof(float));
    conv3d(h_x, h_w, calc_y);
    float diff = 0.0f;
    for (int i = 0; i < size_y; ++i) {
        diff += (h_y[i] - calc_y[i]);
    }
    printf("\n--------diff:%f------\n", diff);

    // Clean up
    free(h_x);
    free(h_w);
    free(h_y);
    free(calc_y);
    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_y);
    cudnnDestroyTensorDescriptor(x_desc);
    cudnnDestroyFilterDescriptor(w_desc);
    cudnnDestroyTensorDescriptor(y_desc);
    cudnnDestroyConvolutionDescriptor(conv_desc);
    cudnnDestroy(cudnn);

    return 0;
}
