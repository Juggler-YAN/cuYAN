#include "hip/hip_runtime.h"
/*
 * group conv
 * C_in 和 C_out 均为 group 的倍数
 */

#include <iostream>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include "../calc/slidingwindow.h"

#define GROUP 4
#define N 6
#define IN_C 8
#define IN_H 7
#define IN_W 7
#define K_H 3
#define K_W 3
#define OUT_C 12
#define OUT_H 3
#define OUT_W 3
#define PAD_H 1
#define PAD_W 1
#define STRIDE_H 2
#define STRIDE_W 2
#define DILATION_H 2
#define DILATION_W 2

void rand_data(float *data, int num, float min, float max) {
    for (int i = 0; i < num; i++) {
        data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() / (float)RAND_MAX) + min);
    }
}

void groupconv(const float* x, const float* w, float* y) {
    for (int n = 0; n < N; ++n) {
        for (int g = 0; g < GROUP; ++g) {
            int xg_size = 1 * IN_C / GROUP * IN_H * IN_W;
            int wg_size = OUT_C / GROUP * IN_C / GROUP * K_H * K_W;
            int yg_size = 1 * OUT_C / GROUP * OUT_H * OUT_W;
            float* xg = (float *)malloc(xg_size * sizeof(float));
            float* wg = (float *)malloc(wg_size * sizeof(float));
            float* yg = (float *)malloc(yg_size * sizeof(float));
            // 第 g 组对应的 x
            memcpy(xg, x + IX(n, g * IN_C / GROUP, 0, 0), xg_size * sizeof(float));
            // 第 g 组对应的 w
            memcpy(wg, w + g * OUT_C / GROUP * IN_C / GROUP * K_H * K_W, wg_size * sizeof(float));
            // conv (1,C_in/group,H_in,W_in) * (C_out/group,C_in,H_k,W_k) = (1,C_out/group,H_out,W_out)
            slidingwindow(xg, wg, yg, 1, IN_C / GROUP, IN_H, IN_W, K_H, K_W, OUT_C / GROUP, OUT_H, OUT_W, PAD_H,
                PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W);
            // 第 g 组对应的 y
            memcpy(y + IY(n, g * OUT_C / GROUP, 0, 0), yg, yg_size * sizeof(float));
            free(xg);
            free(wg);
            free(yg);
        }
    }
}

int main() {
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Initialize cuDNN
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Define the x tensor descriptor
    hipdnnTensorDescriptor_t x_desc;
    hipdnnCreateTensorDescriptor(&x_desc);
    hipdnnSetTensor4dDescriptor(x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, IN_C, IN_H, IN_W);

    // Define the convolution descriptor
    hipdnnFilterDescriptor_t w_desc;
    hipdnnCreateFilterDescriptor(&w_desc);
    hipdnnSetFilter4dDescriptor(w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, OUT_C, IN_C / GROUP, K_H, K_W);

    // Define the convolution descriptor
    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor(conv_desc, PAD_H, PAD_W, STRIDE_H, STRIDE_W, DILATION_H, DILATION_W, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT);

    // Define the y tensor descriptor
    hipdnnTensorDescriptor_t y_desc;
    hipdnnCreateTensorDescriptor(&y_desc);
    hipdnnSetTensor4dDescriptor(y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, OUT_C, OUT_H, OUT_W);

    int size_x = N * IN_C * IN_H * IN_W;
    int size_w = OUT_C / GROUP * IN_C * K_H * K_W;
    int size_y = N * OUT_C * OUT_H * OUT_W;

    // Allocate host memory for x, w, and y
    float *h_x, *h_y, *h_w;
    h_x = (float*)malloc(size_x * sizeof(float));
    h_w = (float*)malloc(size_w * sizeof(float));
    h_y = (float*)malloc(size_y * sizeof(float));

    // Initialization
    rand_data(h_x, size_x, -1, 1);
    rand_data(h_w, size_w, -1, 1);
    rand_data(h_y, size_y, 0, 0);

    // Allocate device memory for x, w, and y
    float *d_x, *d_y, *d_w;
    hipMalloc((void**)&d_x, size_x * sizeof(float));
    hipMalloc((void**)&d_w, size_w * sizeof(float));
    hipMalloc((void**)&d_y, size_y * sizeof(float));

    // Memcpy: host -> device
    hipMemcpy(d_x, h_x, size_x * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size_y * sizeof(float), hipMemcpyHostToDevice);

    // Perform forward convolution
    float alpha = 1.0f, beta = 0.0f;
    // Set the group count
    hipdnnSetConvolutionGroupCount(conv_desc, GROUP);
    hipdnnConvolutionForward(cudnn, &alpha, x_desc, d_x, w_desc, d_w, conv_desc, HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM, nullptr, 0, &beta, y_desc, d_y);

    // Memcpy: device -> host
    hipMemcpy(h_y, d_y, size_y * sizeof(float), hipMemcpyDeviceToHost);

    // Compare
    float *calc_y = (float*)malloc(size_y * sizeof(float));
    groupconv(h_x, h_w, calc_y);
    float diff = 0.0f;
    for (int i = 0; i < size_y; ++i) {
        diff += (h_y[i] - calc_y[i]);
    }
    printf("\n--------diff:%f------\n", diff);

    // Clean up
    free(h_x);
    free(h_w);
    free(h_y);
    free(calc_y);
    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_y);
    hipdnnDestroyTensorDescriptor(x_desc);
    hipdnnDestroyFilterDescriptor(w_desc);
    hipdnnDestroyTensorDescriptor(y_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipdnnDestroy(cudnn);

    return 0;
}
