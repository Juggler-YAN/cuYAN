#include <iostream>
#include <cudnn.h>
#include <hip/hip_runtime.h>

#define N 6
#define IN_C 5
#define IN_L 7
#define K_L 3
#define OUT_C 3
#define OUT_L 3
#define PAD_L 1
#define STRIDE_L 2
#define DILATION_L 2

void rand_data(float *data, int num, float min, float max) {
    for (int i = 0; i < num; i++) {
        data[i] = (fabs(max - min) < 1e-5) ? min : ((max - min) * (rand() / (float)RAND_MAX) + min);
    }
}

void conv1d(const float* x, const float* w, float* y) {

#define IX(n, in_c, in_l) (((n) * IN_C + in_c) * IN_L + in_l)
#define IW(out_c, in_c, k_l) (((out_c) * IN_C + in_c) * K_L + k_l)
#define IY(n, out_c, out_l) (((n) * OUT_C + out_c) * OUT_L + out_l)

    for (int n = 0; n < N; ++n) {
        for (int out_c = 0; out_c < OUT_C; ++out_c) {
            for (int out_l = 0; out_l < OUT_L; ++out_l) {
                float temp = 0.0f;
                for (int k_l = 0; k_l < (DILATION_L - 1) * (K_L - 1) + K_L; k_l += DILATION_L) {
                    int real_in_l = out_l * STRIDE_L + k_l - PAD_L;
                    if (real_in_l >= 0 && real_in_l < IN_L) {
                        int real_k_l = k_l / DILATION_L;
                        for (int in_c = 0; in_c < IN_C; ++in_c) {
                            temp += (float)x[IX(n, in_c, real_in_l)] * (float)w[IW(out_c, in_c, real_k_l)];
                        }
                    }
                }
                y[IY(n, out_c, out_l)] = temp;
            }
        }
    }

}

int main() {
    
    // Initialize CUDA
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA initialization failed!" << std::endl;
        return 1;
    }

    // Initialize cuDNN
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    // Define the x tensor descriptor
    cudnnTensorDescriptor_t x_desc;
    cudnnCreateTensorDescriptor(&x_desc);
    cudnnSetTensor4dDescriptor(x_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, IN_C, IN_L, 1);

    // Define the convolution descriptor
    cudnnFilterDescriptor_t w_desc;
    cudnnCreateFilterDescriptor(&w_desc);
    cudnnSetFilter4dDescriptor(w_desc, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, OUT_C, IN_C, K_L, 1);

    // Define the convolution descriptor
    cudnnConvolutionDescriptor_t conv_desc;
    cudnnCreateConvolutionDescriptor(&conv_desc);
    cudnnSetConvolution2dDescriptor(conv_desc, PAD_L, 0, STRIDE_L, 1, DILATION_L, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);

    // Define the y tensor descriptor
    cudnnTensorDescriptor_t y_desc;
    cudnnCreateTensorDescriptor(&y_desc);
    cudnnSetTensor4dDescriptor(y_desc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, N, OUT_C, OUT_L, 1);

    int size_x = N * IN_C * IN_L;
    int size_w = OUT_C * IN_C * K_L;
    int size_y = N * OUT_C * OUT_L;

    // Allocate host memory for x, w, and y
    float *h_x, *h_y, *h_w;
    h_x = (float*)malloc(size_x * sizeof(float));
    h_w = (float*)malloc(size_w * sizeof(float));
    h_y = (float*)malloc(size_y * sizeof(float));

    // Initialization
    rand_data(h_x, size_x, -1, 1);
    rand_data(h_w, size_w, -1, 1);
    rand_data(h_y, size_y, 0, 0);

    // Allocate device memory for x, w, and y
    float *d_x, *d_y, *d_w;
    hipMalloc((void**)&d_x, size_x * sizeof(float));
    hipMalloc((void**)&d_w, size_w * sizeof(float));
    hipMalloc((void**)&d_y, size_y * sizeof(float));

    // Memcpy: host -> device
    hipMemcpy(d_x, h_x, size_x * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_w, h_w, size_w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size_y * sizeof(float), hipMemcpyHostToDevice);

    // Perform forward convolution
    float alpha = 1.0f, beta = 0.0f;
    cudnnConvolutionForward(cudnn, &alpha, x_desc, d_x, w_desc, d_w, conv_desc, CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM, nullptr, 0, &beta, y_desc, d_y);

    // Memcpy: device -> host
    hipMemcpy(h_y, d_y, size_y * sizeof(float), hipMemcpyDeviceToHost);

    // Compare
    float *calc_y = (float*)malloc(size_y * sizeof(float));
    conv1d(h_x, h_w, calc_y);
    float diff = 0.0f;
    for (int i = 0; i < size_y; ++i) {
        diff += (h_y[i] - calc_y[i]);
    }
    printf("\n--------diff:%f------\n", diff);

    // Clean up
    free(h_x);
    free(h_w);
    free(h_y);
    free(calc_y);
    hipFree(d_x);
    hipFree(d_w);
    hipFree(d_y);
    cudnnDestroyTensorDescriptor(x_desc);
    cudnnDestroyFilterDescriptor(w_desc);
    cudnnDestroyTensorDescriptor(y_desc);
    cudnnDestroyConvolutionDescriptor(conv_desc);
    cudnnDestroy(cudnn);

    return 0;
}
